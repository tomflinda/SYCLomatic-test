// ====------ device_info.cu---------- *- CUDA -* -----------------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===--------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

void test0() {
  // no need to use `cudaSetDevice`
  size_t free_mem, total_mem;
  hipMemGetInfo(&free_mem, &total_mem);

  printf("total_mem : [%lu]\n", total_mem);
  printf("free_mem  : [%lu]\n", free_mem);
}

void test1() {
  hipDeviceProp_t properties;
  hipGetDeviceProperties(&properties, 0);

  const int id = properties.pciDeviceID;
  const hipUUID uuid = properties.uuid;
  int unifiedAddr = properties.unifiedAddressing;
  auto maxTexture1D = properties.maxTexture1D;
  auto maxTexture2D = properties.maxTexture2D;
  auto maxTexture3D = properties.maxTexture3D;
  auto regsPerBlock = properties.regsPerBlock;

  std::cout << "Device ID: " << id << std::endl;
  std::cout << "Device UUID: ";
  for (int i = 0; i < 16; i++) {
    std::cout << std::hex
              << static_cast<int>(static_cast<unsigned char>(uuid.bytes[i]))
              << " ";
  }
  std::cout << std::endl;
  std::cout << "Device maxTexture1D: " << maxTexture1D << std::endl;
  std::cout << "Device maxTexture2D: " << maxTexture2D[0] << " "
            << properties.maxTexture2D[1] << std::endl;
  std::cout << "Device maxTexture3D: " << maxTexture3D[0] << " "
            << properties.maxTexture3D[1] << " " << maxTexture3D[2]
            << std::endl;
  std::cout << "Device regsPerBlock: " << regsPerBlock << std::endl;
}

int main() {
  test0();
  test1();
}
