// ====------ driverTex.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
int main(){
  hipTexRef tex;
  hipDeviceptr_t dptr;
  size_t s, b;
  unsigned int uflag;
  HIPaddress_mode addr_mode;
  HIPfilter_mode filter_mode;
  hipTexRefGetFlags(&uflag, tex);
  hipTexRefGetAddressMode(&addr_mode, tex, 0);
  hipTexRefGetFilterMode(&filter_mode, tex);
  hipTexRefSetAddress(&s, tex, dptr, b);
  HIP_ARRAY_DESCRIPTOR desc;
  hipTexRefSetAddress2D(tex, &desc, dptr, b);
  return 0;
}

