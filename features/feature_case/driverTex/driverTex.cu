// ====------ driverTex.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
int main(){
  hipTexRef tex;
  hipDeviceptr_t dptr;
  size_t s, b;
  unsigned int uflag;
  HIPaddress_mode addr_mode;
  HIPfilter_mode filter_mode;
  hipTexRefGetFlags(&uflag, tex);
  hipTexRefGetAddressMode(&addr_mode, tex, 0);
  hipTexRefGetFilterMode(&filter_mode, tex);
  hipTexRefSetAddress(&s, tex, dptr, b);
  HIP_ARRAY_DESCRIPTOR desc;
  hipTexRefSetAddress2D(tex, &desc, dptr, b);


  hipArray **arr_ptr = new hipArray_t;
  HIP_ARRAY3D_DESCRIPTOR p3DDesc;

  hipArray3DCreate(arr_ptr, &p3DDesc);
  hipArray3DGetDescriptor(&p3DDesc, *arr_ptr);

  HIP_ARRAY_DESCRIPTOR halfDesc;
  hipArrayGetDescriptor(&halfDesc, *arr_ptr);

  cuTexRefCreate(&tex);
  cuTexRefDestroy(tex);
  hipArray_t arr;
  CUsurfref ref;
  cuSurfRefGetArray(&arr, ref);
  cuSurfRefSetArray(ref, arr, 0);
  return 0;
}

