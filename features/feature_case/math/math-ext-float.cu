// ====------------ math-ext-float.cu---------- *- CUDA -* --------------===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===---------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <vector>

using namespace std;

typedef vector<float> f_vector;
typedef pair<float, int> fi_pair;

int passed = 0;
int failed = 0;

void check(bool IsPassed) {
  if (IsPassed) {
    cout << " ---- passed" << endl;
    passed++;
  } else {
    cout << " ---- failed" << endl;
    failed++;
  }
}

template <typename T = float>
void checkResult(const string &FuncName, const vector<T> &Inputs,
                 const float &Expect, const float &DeviceResult,
                 const int precision) {
  cout << FuncName << "(" << Inputs[0];
  for (size_t i = 1; i < Inputs.size(); ++i) {
    cout << ", " << Inputs[i];
  }
  cout << ") = " << fixed << setprecision(precision) << DeviceResult
       << " (expect " << Expect - pow(10, -precision) << " ~ "
       << Expect + pow(10, -precision) << ")";
  cout.unsetf(ios::fixed);
  check(abs(DeviceResult - Expect) < pow(10, -precision));
}

__global__ void cylBesselI0f(float *const Result, float Input1) {
  *Result = cyl_bessel_i0f(Input1);
}

void testCylBesselI0fCases(const vector<pair<float, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    cylBesselI0f<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("cyl_bessel_i0f", {TestCase.first}, TestCase.second.first,
                *Result, TestCase.second.second);
  }
}

__global__ void cylBesselI1f(float *const Result, float Input1) {
  *Result = cyl_bessel_i1f(Input1);
}

void testCylBesselI1fCases(const vector<pair<float, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    cylBesselI1f<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("cyl_bessel_i1f", {TestCase.first}, TestCase.second.first,
                *Result, TestCase.second.second);
  }
}

__global__ void _erfcinvf(float *const DeviceResult, float Input) {
  *DeviceResult = erfcinvf(Input);
}

void testErfcinvf(float *const DeviceResult, float Input) {
  _erfcinvf<<<1, 1>>>(DeviceResult, Input);
  hipDeviceSynchronize();
  // TODO: Need test host side.
}

void testErfcinvfCases(const vector<pair<float, fi_pair>> &TestCases) {
  float *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  // Boundary values.
  testErfcinvf(DeviceResult, 0);
  cout << "erfcinvf(" << 0 << ") = " << *DeviceResult << " (expect inf)";
  check(*DeviceResult > 999999.9);
  testErfcinvf(DeviceResult, 2);
  cout << "erfcinvf(" << 2 << ") = " << *DeviceResult << " (expect -inf)";
  check(*DeviceResult < -999999.9);
  // Other test values.
  for (const auto &TestCase : TestCases) {
    testErfcinvf(DeviceResult, TestCase.first);
    checkResult("erfcinvf", {TestCase.first}, TestCase.second.first,
                *DeviceResult, TestCase.second.second);
  }
}

__global__ void _erfinvf(float *const DeviceResult, float Input) {
  *DeviceResult = erfinvf(Input);
}

void testErfinvf(float *const DeviceResult, float Input) {
  _erfinvf<<<1, 1>>>(DeviceResult, Input);
  hipDeviceSynchronize();
  // Call from host.
}

void testErfinvfCases(const vector<pair<float, fi_pair>> &TestCases) {
  float *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  // Boundary values.
  testErfinvf(DeviceResult, -1);
  cout << "erfinvf(" << -1 << ") = " << *DeviceResult << " (expect -inf)";
  check(*DeviceResult < -999999.9);
  testErfinvf(DeviceResult, 1);
  cout << "erfinvf(" << 1 << ") = " << *DeviceResult << " (expect inf)";
  check(*DeviceResult > 999999.9);
  // Other test values.
  for (const auto &TestCase : TestCases) {
    testErfinvf(DeviceResult, TestCase.first);
    checkResult("erfinvf", {TestCase.first}, TestCase.second.first,
                *DeviceResult, TestCase.second.second);
  }
}

__global__ void _j0f(float *const Result, float Input1) {
  *Result = j0f(Input1);
}

void testJ0fCases(const vector<pair<float, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _j0f<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("j0f", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _j1f(float *const Result, float Input1) {
  *Result = j1f(Input1);
}

void testJ1fCases(const vector<pair<float, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _j1f<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("j1f", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _jnf(float *const Result, int Input1, float Input2) {
  *Result = jnf(Input1, Input2);
}

void testJnfCases(const vector<pair<pair<int, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _jnf<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("jnf", {(float)TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void _normcdff(float *const DeviceResult, float Input) {
  *DeviceResult = normcdff(Input);
}

void testNormcdff(float *const DeviceResult, float Input) {
  _normcdff<<<1, 1>>>(DeviceResult, Input);
  hipDeviceSynchronize();
  // Call from host.
}

void testNormcdffCases(const vector<pair<float, fi_pair>> &TestCases) {
  float *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  for (const auto &TestCase : TestCases) {
    testNormcdff(DeviceResult, TestCase.first);
    checkResult("normcdff", {TestCase.first}, TestCase.second.first,
                *DeviceResult, TestCase.second.second);
  }
}

__global__ void _normcdfinvf(float *const DeviceResult, float Input) {
  *DeviceResult = normcdfinvf(Input);
}

void testNormcdfinvf(float *const DeviceResult, float Input) {
  _normcdfinvf<<<1, 1>>>(DeviceResult, Input);
  hipDeviceSynchronize();
  // Call from host.
}

void testNormcdfinvfCases(const vector<pair<float, fi_pair>> &TestCases) {
  float *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  // Boundary values.
  testNormcdfinvf(DeviceResult, 0);
  cout << "normcdfinvf(" << 0 << ") = " << *DeviceResult << " (expect -inf)";
  check(*DeviceResult < -999999.9);
  testNormcdfinvf(DeviceResult, 1);
  cout << "normcdfinvf(" << 1 << ") = " << *DeviceResult << " (expect inf)";
  check(*DeviceResult > 999999.9);
  // Other test values.
  for (const auto &TestCase : TestCases) {
    testNormcdfinvf(DeviceResult, TestCase.first);
    checkResult("normcdfinvf", {TestCase.first}, TestCase.second.first,
                *DeviceResult, TestCase.second.second);
  }
}

__global__ void setVecValue(float *Input1, const float Input2) {
  *Input1 = Input2;
}

__global__ void _normf(float *const DeviceResult, int Input1,
                       const float *Input2) {
  *DeviceResult = normf(Input1, Input2);
}

void testNormf(float *const DeviceResult, int Input1, const float *Input2) {
  _normf<<<1, 1>>>(DeviceResult, Input1, Input2);
  hipDeviceSynchronize();
  // Call from host.
}

void testNormfCases(const vector<pair<f_vector, fi_pair>> &TestCases) {
  float *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  for (const auto &TestCase : TestCases) {
    float *Input;
    hipMallocManaged(&Input, TestCase.first.size() * sizeof(*Input));
    for (size_t i = 0; i < TestCase.first.size(); ++i) {
      // Notice: cannot set value from host!
      setVecValue<<<1, 1>>>(Input + i, TestCase.first[i]);
      hipDeviceSynchronize();
    }
    testNormf(DeviceResult, TestCase.first.size(), Input);
    string arg = "&{";
    for (size_t i = 0; i < TestCase.first.size() - 1; ++i) {
      arg += to_string(TestCase.first[i]) + ", ";
    }
    arg += to_string(TestCase.first.back()) + "}";
    checkResult<string>("normf", {to_string(TestCase.first.size()), arg},
                        TestCase.second.first, *DeviceResult,
                        TestCase.second.second);
  }
}

__global__ void _rnormf(float *const DeviceResult, int Input1,
                        const float *Input2) {
  *DeviceResult = rnormf(Input1, Input2);
}

void testRnormf(float *const DeviceResult, int Input1, const float *Input2) {
  _rnormf<<<1, 1>>>(DeviceResult, Input1, Input2);
  hipDeviceSynchronize();
  // Call from host.
}

void testRnormfCases(const vector<pair<f_vector, fi_pair>> &TestCases) {
  float *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  for (const auto &TestCase : TestCases) {
    float *Input;
    hipMallocManaged(&Input, TestCase.first.size() * sizeof(*Input));
    for (size_t i = 0; i < TestCase.first.size(); ++i) {
      // Notice: cannot set value from host!
      setVecValue<<<1, 1>>>(Input + i, TestCase.first[i]);
      hipDeviceSynchronize();
    }
    testRnormf(DeviceResult, TestCase.first.size(), Input);
    string arg = "&{";
    for (size_t i = 0; i < TestCase.first.size() - 1; ++i) {
      arg += to_string(TestCase.first[i]) + ", ";
    }
    arg += to_string(TestCase.first.back()) + "}";
    checkResult<string>("rnormf", {to_string(TestCase.first.size()), arg},
                        TestCase.second.first, *DeviceResult,
                        TestCase.second.second);
  }
}

__global__ void _y0f(float *const Result, float Input1) {
  *Result = y0f(Input1);
}

void testY0fCases(const vector<pair<float, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _y0f<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("y0f", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _y1f(float *const Result, float Input1) {
  *Result = y1f(Input1);
}

void testY1fCases(const vector<pair<float, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _y1f<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("y1f", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _ynf(float *const Result, int Input1, float Input2) {
  *Result = ynf(Input1, Input2);
}

void testYnfCases(const vector<pair<pair<int, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _ynf<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("ynf", {(float)TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

// Single Precision Intrinsics

__global__ void fadd_rd(float *const Result, float Input1, float Input2) {
  *Result = __fadd_rd(Input1, Input2);
}

void testFadd_rdCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fadd_rd<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fadd_rd", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fadd_rn(float *const Result, float Input1, float Input2) {
  *Result = __fadd_rn(Input1, Input2);
}

void testFadd_rnCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fadd_rn<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fadd_rn", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fadd_ru(float *const Result, float Input1, float Input2) {
  *Result = __fadd_ru(Input1, Input2);
}

void testFadd_ruCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fadd_ru<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fadd_ru", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fadd_rz(float *const Result, float Input1, float Input2) {
  *Result = __fadd_rz(Input1, Input2);
}

void testFadd_rzCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fadd_rz<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fadd_rz", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fmaf_rd(float *const Result, float Input1, float Input2,
                        float Input3) {
  *Result = __fmaf_rd(Input1, Input2, Input3);
}

void testFmaf_rdCases(const vector<pair<vector<float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmaf_rd<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                      TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fmaf_rd", TestCase.first, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void fmaf_rn(float *const Result, float Input1, float Input2,
                        float Input3) {
  *Result = __fmaf_rn(Input1, Input2, Input3);
}

void testFmaf_rnCases(const vector<pair<vector<float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmaf_rn<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                      TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fmaf_rn", TestCase.first, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void fmaf_ru(float *const Result, float Input1, float Input2,
                        float Input3) {
  *Result = __fmaf_ru(Input1, Input2, Input3);
}

void testFmaf_ruCases(const vector<pair<vector<float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmaf_ru<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                      TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fmaf_ru", TestCase.first, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void fmaf_rz(float *const Result, float Input1, float Input2,
                        float Input3) {
  *Result = __fmaf_rz(Input1, Input2, Input3);
}

void testFmaf_rzCases(const vector<pair<vector<float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmaf_rz<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                      TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fmaf_rz", TestCase.first, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void fmaf_ieee_rd(float *const Result, float Input1, float Input2,
                             float Input3) {
  *Result = __fmaf_ieee_rd(Input1, Input2, Input3);
}

void testFmaf_ieee_rdCases(
    const vector<pair<vector<float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmaf_ieee_rd<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                           TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fmaf_ieee_rd", TestCase.first, TestCase.second.first,
                *Result, TestCase.second.second);
  }
}

__global__ void fmaf_ieee_rn(float *const Result, float Input1, float Input2,
                             float Input3) {
  *Result = __fmaf_ieee_rn(Input1, Input2, Input3);
}

void testFmaf_ieee_rnCases(
    const vector<pair<vector<float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmaf_ieee_rn<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                           TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fmaf_ieee_rn", TestCase.first, TestCase.second.first,
                *Result, TestCase.second.second);
  }
}

__global__ void fmaf_ieee_ru(float *const Result, float Input1, float Input2,
                             float Input3) {
  *Result = __fmaf_ieee_ru(Input1, Input2, Input3);
}

void testFmaf_ieee_ruCases(
    const vector<pair<vector<float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmaf_ieee_ru<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                           TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fmaf_ieee_ru", TestCase.first, TestCase.second.first,
                *Result, TestCase.second.second);
  }
}

__global__ void fmaf_ieee_rz(float *const Result, float Input1, float Input2,
                             float Input3) {
  *Result = __fmaf_ieee_rz(Input1, Input2, Input3);
}

void testFmaf_ieee_rzCases(
    const vector<pair<vector<float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmaf_ieee_rz<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                           TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fmaf_ieee_rz", TestCase.first, TestCase.second.first,
                *Result, TestCase.second.second);
  }
}

__global__ void fmul_rd(float *const Result, float Input1, float Input2) {
  *Result = __fmul_rd(Input1, Input2);
}

void testFmul_rdCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmul_rd<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fmul_rd", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fmul_rn(float *const Result, float Input1, float Input2) {
  *Result = __fmul_rn(Input1, Input2);
}

void testFmul_rnCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmul_rn<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fmul_rn", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fmul_ru(float *const Result, float Input1, float Input2) {
  *Result = __fmul_ru(Input1, Input2);
}

void testFmul_ruCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmul_ru<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fmul_ru", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fmul_rz(float *const Result, float Input1, float Input2) {
  *Result = __fmul_rz(Input1, Input2);
}

void testFmul_rzCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmul_rz<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fmul_rz", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fsub_rd(float *const Result, float Input1, float Input2) {
  *Result = __fsub_rd(Input1, Input2);
}

void testFsub_rdCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fsub_rd<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fsub_rd", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fsub_rn(float *const Result, float Input1, float Input2) {
  *Result = __fsub_rn(Input1, Input2);
}

void testFsub_rnCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fsub_rn<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fsub_rn", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fsub_ru(float *const Result, float Input1, float Input2) {
  *Result = __fsub_ru(Input1, Input2);
}

void testFsub_ruCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fsub_ru<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fsub_ru", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fsub_rz(float *const Result, float Input1, float Input2) {
  *Result = __fsub_rz(Input1, Input2);
}

void testFsub_rzCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fsub_rz<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fsub_rz", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

int main() {
  testCylBesselI0fCases({
      {0.3, {1.022626876831055, 15}},
      {0.5, {1.063483357429504, 15}},
      {0.8, {1.166514992713928, 15}},
      {1.6, {1.749980688095093, 15}},
      {-5, {27.23987197875977, 14}},
  });
  testCylBesselI1fCases({
      {0.3, {0.1516939, 7}},
      {0.5, {0.2578943073749542, 16}},
      {0.8, {0.4328648149967194, 16}},
      {1.6, {1.084811, 6}},
      {-5, {-24.33564186096191, 14}},
  });
  testErfcinvfCases({
      {0.3, {0.7328690886497498, 16}},
      {0.5, {0.4769362807273865, 16}},
      {0.8, {0.1791434437036514, 16}},
      {1.6, {-0.5951161, 7}},
  });
  testErfinvfCases({
      {-0.3, {-0.2724627256393433, 16}},
      {-0.5, {-0.4769362807273865, 16}},
      {0, {0, 37}},
      {0.5, {0.4769362807273865, 16}},
  });
  testJ0fCases({
      {0.3, {0.9776262, 7}},
      {0.5, {0.9384698271751404, 16}},
      {0.8, {0.8462873, 7}},
      {1.6, {0.4554022, 7}},
      {-5, {-0.1775968, 7}},
  });
  testJ1fCases({
      {0.3, {0.1483188, 7}},
      {0.5, {0.2422684580087662, 16}},
      {0.8, {0.3688420653343201, 16}},
      {1.6, {0.569896, 7}},
      {-5, {0.3275791406631470, 16}},
  });
  testJnfCases({
      {{1, 0.3}, {0.1483188, 7}},
      {{2, 0.5}, {0.03060402534902096, 17}},
      {{3, 0.8}, {0.010246766731142998, 18}},
      {{4, 1.6}, {0.014995161, 9}},
      {{5, -5}, {-0.2611406, 7}},
  });
  testNormcdffCases({
      {-5, {0.0000002866515842470108, 22}},
      {-3, {0.001349898055195808, 18}},
      {0, {0.5, 16}},
      {1, {0.8413447141647339, 16}},
      {5, {0.9999997019767761, 16}},
  });
  testNormcdfinvfCases({
      {0.3, {-0.5244004130363464, 16}},
      {0.5, {0, 37}},
      {0.8, {0.8416212, 7}},
  });
  testNormfCases({
      {{-0.3, -0.34, -0.98}, {1.079814791679382, 15}},
      {{0.3, 0.34, 0.98}, {1.079814791679382, 15}},
      {{0.5}, {0.5, 16}},
      {{23, 432, 23, 456, 23}, {629.402099609375, 13}},
  });
  testRnormfCases({
      {{-0.3, -0.34, -0.98}, {0.9260847, 7}},
      {{0.3, 0.34, 0.98}, {0.9260847, 7}},
      {{0.5}, {2, 15}},
      {{23, 432, 23, 456, 23}, {0.001588809420354664, 18}},
  });
  testY0fCases({
      {0.3, {-0.8072735, 7}},
      {0.5, {-0.4445187, 7}},
      {0.8, {-0.08680226, 8}},
      {1.6, {0.4204270, 7}},
      {5, {-0.3085176050662994, 16}},
  });
  testY1fCases({
      {0.3, {-2.293104887008667, 15}},
      {0.5, {-1.471472, 6}},
      {0.8, {-0.9781441, 7}},
      {1.6, {-0.3475780, 7}},
      {5, {0.1478631347417831, 16}},
  });
  testYnfCases({
      {{1, 0.3}, {-2.293104887008667, 15}},
      {{2, 0.5}, {-5.441371, 6}},
      {{3, 0.8}, {-10.814646, 5}},
      {{4, 1.6}, {-5.856365, 6}},
      {{0, 5}, {-0.3085176050662994, 16}},
  });
  testFadd_rdCases({
      {{-0.3, -0.4}, {-0.7000000476837158, 16}},
      {{0.3, -0.4}, {-0.09999999403953552, 17}},
      {{0.3, 0.4}, {0.699999988079071, 16}},
      {{0.3, 0.8}, {1.100000023841858, 15}},
      {{3, 4}, {7, 15}},
  });
  testFadd_rnCases({
      {{-0.3, -0.4}, {-0.7000000476837158, 16}},
      {{0.3, -0.4}, {-0.09999999403953552, 17}},
      {{0.3, 0.4}, {0.7000000476837158, 16}},
      {{0.3, 0.8}, {1.100000023841858, 15}},
      {{3, 4}, {7, 15}},
  });
  testFadd_ruCases({
      {{-0.3, -0.4}, {-0.699999988079071, 16}},
      {{0.3, -0.4}, {-0.09999999403953552, 17}},
      {{0.3, 0.4}, {0.7000000476837158, 16}},
      {{0.3, 0.8}, {1.100000023841858, 15}},
      {{3, 4}, {7, 15}},
  });
  testFadd_rzCases({
      {{-0.3, -0.4}, {-0.699999988079071, 16}},
      {{0.3, -0.4}, {-0.09999999403953552, 17}},
      {{0.3, 0.4}, {0.699999988079071, 16}},
      {{0.3, 0.8}, {1.100000023841858, 15}},
      {{3, 4}, {7, 15}},
  });
  testFmaf_rdCases({
      {{-0.3, -0.4, -0.2}, {-0.07999999821186066, 17}},
      {{0.3, -0.4, -0.1}, {-0.2200000137090683, 16}},
      {{0.3, 0.4, 0.1}, {0.2199999988079071, 16}},
      {{0.3, 0.4, 0}, {0.12000000476837158, 17}},
      {{3, 4, 5}, {17, 14}},
  });
  testFmaf_rnCases({
      {{-0.3, -0.4, -0.2}, {-0.07999999821186066, 17}},
      {{0.3, -0.4, -0.1}, {-0.2200000137090683, 16}},
      {{0.3, 0.4, 0.1}, {0.2200000137090683, 16}},
      {{0.3, 0.4, 0}, {0.12000000476837158, 17}},
      {{3, 4, 5}, {17, 14}},
  });
  testFmaf_ruCases({
      {{-0.3, -0.4, -0.2}, {-0.07999999076128006, 17}},
      {{0.3, -0.4, -0.1}, {-0.2199999988079071, 16}},
      {{0.3, 0.4, 0.1}, {0.2200000137090683, 16}},
      {{0.3, 0.4, 0}, {0.12000001221895218, 17}},
      {{3, 4, 5}, {17, 14}},
  });
  testFmaf_rzCases({
      {{-0.3, -0.4, -0.2}, {-0.07999999076128006, 17}},
      {{0.3, -0.4, -0.1}, {-0.2199999988079071, 16}},
      {{0.3, 0.4, 0.1}, {0.2199999988079071, 16}},
      {{0.3, 0.4, 0}, {0.12000000476837158, 17}},
      {{3, 4, 5}, {17, 14}},
  });
  testFmaf_ieee_rdCases({
      {{-0.3, -0.4, -0.2}, {-0.07999999821186066, 17}},
      {{0.3, -0.4, -0.1}, {-0.2200000137090683, 16}},
      {{0.3, 0.4, 0.1}, {0.2199999988079071, 16}},
      {{0.3, 0.4, 0}, {0.12000000476837158, 17}},
      {{3, 4, 5}, {17, 14}},
  });
  testFmaf_ieee_rnCases({
      {{-0.3, -0.4, -0.2}, {-0.07999999821186066, 17}},
      {{0.3, -0.4, -0.1}, {-0.2200000137090683, 16}},
      {{0.3, 0.4, 0.1}, {0.2200000137090683, 16}},
      {{0.3, 0.4, 0}, {0.12000000476837158, 17}},
      {{3, 4, 5}, {17, 14}},
  });
  testFmaf_ieee_ruCases({
      {{-0.3, -0.4, -0.2}, {-0.07999999076128006, 17}},
      {{0.3, -0.4, -0.1}, {-0.2199999988079071, 16}},
      {{0.3, 0.4, 0.1}, {0.2200000137090683, 16}},
      {{0.3, 0.4, 0}, {0.12000001221895218, 17}},
      {{3, 4, 5}, {17, 14}},
  });
  testFmaf_ieee_rzCases({
      {{-0.3, -0.4, -0.2}, {-0.07999999076128006, 17}},
      {{0.3, -0.4, -0.1}, {-0.2199999988079071, 16}},
      {{0.3, 0.4, 0.1}, {0.2199999988079071, 16}},
      {{0.3, 0.4, 0}, {0.12000000476837158, 17}},
      {{3, 4, 5}, {17, 14}},
  });
  testFmul_rdCases({
      {{-0.3, -0.4}, {0.12000000476837158, 17}},
      {{0.3, -0.4}, {-0.12000001221895218, 17}},
      {{0.3, 0.4}, {0.12000000476837158, 17}},
      {{0.3, 0.8}, {0.2400000095367432, 16}},
      {{3, 4}, {12, 15}},
  });
  testFmul_rnCases({
      {{-0.3, -0.4}, {0.12000000476837158, 17}},
      {{0.3, -0.4}, {-0.12000000476837158, 17}},
      {{0.3, 0.4}, {0.12000000476837158, 17}},
      {{0.3, 0.8}, {0.2400000095367432, 16}},
      {{3, 4}, {12, 15}},
  });
  testFmul_ruCases({
      {{-0.3, -0.4}, {0.12000001221895218, 17}},
      {{0.3, -0.4}, {-0.12000000476837158, 17}},
      {{0.3, 0.4}, {0.12000001221895218, 17}},
      {{0.3, 0.8}, {0.2400000244379044, 16}},
      {{3, 4}, {12, 15}},
  });
  testFmul_rzCases({
      {{-0.3, -0.4}, {0.12000000476837158, 17}},
      {{0.3, -0.4}, {-0.12000000476837158, 17}},
      {{0.3, 0.4}, {0.12000000476837158, 17}},
      {{0.3, 0.8}, {0.2400000095367432, 16}},
      {{3, 4}, {12, 15}},
  });
  testFsub_rdCases({
      {{-0.3, -0.4}, {0.09999999403953552, 17}},
      {{0.3, -0.4}, {0.699999988079071, 16}},
      {{0.3, 0.4}, {-0.09999999403953552, 17}},
      {{0.3, 0.8}, {-0.5, 16}},
      {{3, 4}, {-1, 15}},
  });
  testFsub_rnCases({
      {{-0.3, -0.4}, {0.09999999403953552, 17}},
      {{0.3, -0.4}, {0.7000000476837158, 16}},
      {{0.3, 0.4}, {-0.09999999403953552, 17}},
      {{0.3, 0.8}, {-0.5, 16}},
      {{3, 4}, {-1, 15}},
  });
  testFsub_ruCases({
      {{-0.3, -0.4}, {0.09999999403953552, 17}},
      {{0.3, -0.4}, {0.7000000476837158, 16}},
      {{0.3, 0.4}, {-0.09999999403953552, 17}},
      {{0.3, 0.8}, {-0.5, 16}},
      {{3, 4}, {-1, 15}},
  });
  testFsub_rzCases({
      {{-0.3, -0.4}, {0.09999999403953552, 17}},
      {{0.3, -0.4}, {0.699999988079071, 16}},
      {{0.3, 0.4}, {-0.09999999403953552, 17}},
      {{0.3, 0.8}, {-0.5, 16}},
      {{3, 4}, {-1, 15}},
  });
  cout << "passed " << passed << "/" << passed + failed << " cases!" << endl;
  if (failed) {
    cout << "failed!" << endl;
  }
  return failed;
}
