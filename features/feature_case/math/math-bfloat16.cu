// ====-------------- math-bfloat16.cu---------- *- CUDA -* -------------===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===---------------------------------------------------------------------===//

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <iostream>

__device__ uint16_t convertToU16(__hip_bfloat16 value) {
  union {
    __hip_bfloat16 bf16;
    uint16_t u16;
  } TypePun;
  TypePun.bf16 = value;
  return TypePun.u16;
}

__device__ bool valuesAreClose(float a, float b, float epsilon = 0.05f) {
  return (fabsf(a - b) < epsilon);
}

__global__ void testMathFunctions(char *const TestResults) {
  const __hip_bfloat16 bf16 = __float2bfloat16(3.14f);
  const float f32 = __bfloat162float(bf16);

  // Check that the intermediate bfloat16 value has the expected byte
  // representation. The CUDA and SYCL values both use round-to-nearest-even
  // rounding mode.
  TestResults[0] = (convertToU16(bf16) == 0x4049);

  // Check that the converted value is close to the original. The two values
  // may differ slightly due to the loss of precision during type conversion.
  TestResults[1] = valuesAreClose(f32, 3.14f);
}

int main() {
  constexpr int NumberOfTests = 2;
  char *TestResults;
  hipMallocManaged(&TestResults, NumberOfTests * sizeof(*TestResults));
  testMathFunctions<<<1, 1>>>(TestResults);
  hipDeviceSynchronize();
  for (int i = 0; i < NumberOfTests; i++) {
    if (TestResults[i] == 0) {
      std::cout << "Test " << i << " failed" << std::endl;
      return 1;
    }
  }
  return 0;
}
