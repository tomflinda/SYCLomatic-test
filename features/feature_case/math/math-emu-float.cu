// ===-------------- math-emu-float.cu---------- *- CUDA -* ---------------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===---------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <vector>

using namespace std;

typedef vector<float> f_vector;
typedef tuple<float, float, float> f_tuple3;
typedef tuple<float, float, float, float> f_tuple4;
typedef pair<float, int> fi_pair;

int passed = 0;
int failed = 0;

void check(bool IsPassed) {
  if (IsPassed) {
    cout << " ---- passed" << endl;
    passed++;
  } else {
    cout << " ---- failed" << endl;
    failed++;
  }
}

template <typename T = float>
void checkResult(const string &FuncName, const vector<T> &Inputs,
                 const float &Expect, const float &DeviceResult,
                 const int precision) {
  cout << FuncName << "(" << Inputs[0];
  for (size_t i = 1; i < Inputs.size(); ++i) {
    cout << ", " << Inputs[i];
  }
  cout << ") = " << fixed << setprecision(precision < 0 ? 0 : precision)
       << DeviceResult << " (expect " << Expect - pow(10, -precision) << " ~ "
       << Expect + pow(10, -precision) << ")";
  cout.unsetf(ios::fixed);
  check(abs(DeviceResult - Expect) < pow(10, -precision));
}

// Single Precision Mathematical Functions

__global__ void expf(float *const Result, float Input1) {
  *Result = expf(Input1);
}

void testExpfCases(const vector<pair<float, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    expf<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("expf", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _norm3df(float *const DeviceResult, float Input1, float Input2,
                         float Input3) {
  *DeviceResult = norm3df(Input1, Input2, Input3);
}

void testNorm3df(float *const DeviceResult, float Input1, float Input2,
                 float Input3) {
  _norm3df<<<1, 1>>>(DeviceResult, Input1, Input2, Input3);
  hipDeviceSynchronize();
  // TODO: Need test host side.
}

void testNorm3dfCases(const vector<pair<f_tuple3, fi_pair>> &TestCases) {
  float *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  for (const auto &TestCase : TestCases) {
    testNorm3df(DeviceResult, get<0>(TestCase.first), get<1>(TestCase.first),
                get<2>(TestCase.first));
    checkResult("norm3df",
                {get<0>(TestCase.first), get<1>(TestCase.first),
                 get<2>(TestCase.first)},
                TestCase.second.first, *DeviceResult, TestCase.second.second);
  }
}

__global__ void _norm4d(float *const DeviceResult, float Input1, float Input2,
                        float Input3, float Input4) {
  *DeviceResult = norm4df(Input1, Input2, Input3, Input4);
}

void testNorm4df(float *const DeviceResult, float Input1, float Input2,
                 float Input3, float Input4) {
  _norm4d<<<1, 1>>>(DeviceResult, Input1, Input2, Input3, Input4);
  hipDeviceSynchronize();
  // Call from host.
}

void testNorm4dfCases(const vector<pair<f_tuple4, fi_pair>> &TestCases) {
  float *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  for (const auto &TestCase : TestCases) {
    testNorm4df(DeviceResult, get<0>(TestCase.first), get<1>(TestCase.first),
                get<2>(TestCase.first), get<3>(TestCase.first));
    checkResult("norm4d",
                {get<0>(TestCase.first), get<1>(TestCase.first),
                 get<2>(TestCase.first), get<3>(TestCase.first)},
                TestCase.second.first, *DeviceResult, TestCase.second.second);
  }
}

__global__ void _normcdff(float *const DeviceResult, float Input) {
  *DeviceResult = normcdff(Input);
}

void testNormcdff(float *const DeviceResult, float Input) {
  _normcdff<<<1, 1>>>(DeviceResult, Input);
  hipDeviceSynchronize();
  // Call from host.
}

void testNormcdffCases(const vector<pair<float, fi_pair>> &TestCases) {
  float *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  // Other test values.
  for (const auto &TestCase : TestCases) {
    testNormcdff(DeviceResult, TestCase.first);
    checkResult("normcdff", {TestCase.first}, TestCase.second.first,
                *DeviceResult, TestCase.second.second);
  }
}

__global__ void setVecValue(float *Input1, const float Input2) {
  *Input1 = Input2;
}

__global__ void _normf(float *const DeviceResult, int Input1,
                       const float *Input2) {
  *DeviceResult = normf(Input1, Input2);
}

void testNormf(float *const DeviceResult, int Input1, const float *Input2) {
  _normf<<<1, 1>>>(DeviceResult, Input1, Input2);
  hipDeviceSynchronize();
  // Call from host.
}

void testNormfCases(const vector<pair<f_vector, fi_pair>> &TestCases) {
  float *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  // Other test values.
  for (const auto &TestCase : TestCases) {
    float *Input;
    hipMallocManaged(&Input, TestCase.first.size() * sizeof(*Input));
    for (size_t i = 0; i < TestCase.first.size(); ++i) {
      // Notice: cannot set value from host!
      setVecValue<<<1, 1>>>(Input + i, TestCase.first[i]);
      hipDeviceSynchronize();
    }
    testNormf(DeviceResult, TestCase.first.size(), Input);
    string arg = "&{";
    for (size_t i = 0; i < TestCase.first.size() - 1; ++i) {
      arg += to_string(TestCase.first[i]) + ", ";
    }
    arg += to_string(TestCase.first.back()) + "}";
    checkResult<string>("normf", {to_string(TestCase.first.size()), arg},
                        TestCase.second.first, *DeviceResult,
                        TestCase.second.second);
  }
}

__global__ void _rcbrtf(float *const DeviceResult, float Input1) {
  *DeviceResult = rcbrtf(Input1);
}

void testRcbrtfCases(const vector<pair<float, fi_pair>> &TestCases) {
  float *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  for (const auto &TestCase : TestCases) {
    _rcbrtf<<<1, 1>>>(DeviceResult, TestCase.first);
    hipDeviceSynchronize();
    checkResult("rcbrtf", {TestCase.first}, TestCase.second.first,
                *DeviceResult, TestCase.second.second);
  }
}

__global__ void _rnorm3df(float *const DeviceResult, float Input1, float Input2,
                          float Input3) {
  *DeviceResult = rnorm3df(Input1, Input2, Input3);
}

void testRnorm3df(float *const DeviceResult, float Input1, float Input2,
                  float Input3) {
  _rnorm3df<<<1, 1>>>(DeviceResult, Input1, Input2, Input3);
  hipDeviceSynchronize();
  // Call from host.
}

void testRnorm3dfCases(const vector<pair<f_tuple3, fi_pair>> &TestCases) {
  float *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  for (const auto &TestCase : TestCases) {
    testRnorm3df(DeviceResult, get<0>(TestCase.first), get<1>(TestCase.first),
                 get<2>(TestCase.first));
    checkResult("rnorm3df",
                {get<0>(TestCase.first), get<1>(TestCase.first),
                 get<2>(TestCase.first)},
                TestCase.second.first, *DeviceResult, TestCase.second.second);
  }
}

__global__ void _rnorm4df(float *const DeviceResult, float Input1, float Input2,
                          float Input3, float Input4) {
  *DeviceResult = rnorm4df(Input1, Input2, Input3, Input4);
}

void testRnorm4df(float *const DeviceResult, float Input1, float Input2,
                  float Input3, float Input4) {
  _rnorm4df<<<1, 1>>>(DeviceResult, Input1, Input2, Input3, Input4);
  hipDeviceSynchronize();
  // Call from host.
}

void testRnorm4dfCases(const vector<pair<f_tuple4, fi_pair>> &TestCases) {
  float *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  for (const auto &TestCase : TestCases) {
    testRnorm4df(DeviceResult, get<0>(TestCase.first), get<1>(TestCase.first),
                 get<2>(TestCase.first), get<3>(TestCase.first));
    checkResult("rnorm4df",
                {get<0>(TestCase.first), get<1>(TestCase.first),
                 get<2>(TestCase.first), get<3>(TestCase.first)},
                TestCase.second.first, *DeviceResult, TestCase.second.second);
  }
}

__global__ void _rnormf(float *const DeviceResult, int Input1,
                        const float *Input2) {
  *DeviceResult = rnormf(Input1, Input2);
}

void testRnormf(float *const DeviceResult, int Input1, const float *Input2) {
  _rnormf<<<1, 1>>>(DeviceResult, Input1, Input2);
  hipDeviceSynchronize();
  // Call from host.
}

void testRnormfCases(const vector<pair<f_vector, fi_pair>> &TestCases) {
  float *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  // Other test values.
  for (const auto &TestCase : TestCases) {
    float *Input;
    hipMallocManaged(&Input, TestCase.first.size() * sizeof(*Input));
    for (size_t i = 0; i < TestCase.first.size(); ++i) {
      // Notice: cannot set value from host!
      setVecValue<<<1, 1>>>(Input + i, TestCase.first[i]);
      hipDeviceSynchronize();
    }
    testRnormf(DeviceResult, TestCase.first.size(), Input);
    string arg = "&{";
    for (size_t i = 0; i < TestCase.first.size() - 1; ++i) {
      arg += to_string(TestCase.first[i]) + ", ";
    }
    arg += to_string(TestCase.first.back()) + "}";
    checkResult<string>("rnormf", {to_string(TestCase.first.size()), arg},
                        TestCase.second.first, *DeviceResult,
                        TestCase.second.second);
  }
}

// Single Precision Intrinsics

__global__ void _expf(float *const Result, float Input1) {
  *Result = __expf(Input1);
}

void test_ExpfCases(const vector<pair<float, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _expf<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("__expf", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void fadd_rd(float *const Result, float Input1, float Input2) {
  *Result = __fadd_rd(Input1, Input2);
}

void testFadd_rdCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fadd_rd<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fadd_rd", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fadd_rn(float *const Result, float Input1, float Input2) {
  *Result = __fadd_rn(Input1, Input2);
}

void testFadd_rnCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fadd_rn<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fadd_rn", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fadd_ru(float *const Result, float Input1, float Input2) {
  *Result = __fadd_ru(Input1, Input2);
}

void testFadd_ruCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fadd_ru<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fadd_ru", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fadd_rz(float *const Result, float Input1, float Input2) {
  *Result = __fadd_rz(Input1, Input2);
}

void testFadd_rzCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fadd_rz<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fadd_rz", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fmaf_rd(float *const Result, float Input1, float Input2,
                        float Input3) {
  *Result = __fmaf_rd(Input1, Input2, Input3);
}

void testFmaf_rdCases(const vector<pair<vector<float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmaf_rd<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                      TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fmaf_rd", TestCase.first, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void fmaf_rn(float *const Result, float Input1, float Input2,
                        float Input3) {
  *Result = __fmaf_rn(Input1, Input2, Input3);
}

void testFmaf_rnCases(const vector<pair<vector<float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmaf_rn<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                      TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fmaf_rn", TestCase.first, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void fmaf_ru(float *const Result, float Input1, float Input2,
                        float Input3) {
  *Result = __fmaf_ru(Input1, Input2, Input3);
}

void testFmaf_ruCases(const vector<pair<vector<float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmaf_ru<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                      TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fmaf_ru", TestCase.first, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void fmaf_rz(float *const Result, float Input1, float Input2,
                        float Input3) {
  *Result = __fmaf_rz(Input1, Input2, Input3);
}

void testFmaf_rzCases(const vector<pair<vector<float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmaf_rz<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                      TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fmaf_rz", TestCase.first, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void fmaf_ieee_rd(float *const Result, float Input1, float Input2,
                             float Input3) {
  *Result = __fmaf_ieee_rd(Input1, Input2, Input3);
}

void testFmaf_ieee_rdCases(
    const vector<pair<vector<float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmaf_ieee_rd<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                           TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fmaf_ieee_rd", TestCase.first, TestCase.second.first,
                *Result, TestCase.second.second);
  }
}

__global__ void fmaf_ieee_rn(float *const Result, float Input1, float Input2,
                             float Input3) {
  *Result = __fmaf_ieee_rn(Input1, Input2, Input3);
}

void testFmaf_ieee_rnCases(
    const vector<pair<vector<float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmaf_ieee_rn<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                           TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fmaf_ieee_rn", TestCase.first, TestCase.second.first,
                *Result, TestCase.second.second);
  }
}

__global__ void fmaf_ieee_ru(float *const Result, float Input1, float Input2,
                             float Input3) {
  *Result = __fmaf_ieee_ru(Input1, Input2, Input3);
}

void testFmaf_ieee_ruCases(
    const vector<pair<vector<float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmaf_ieee_ru<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                           TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fmaf_ieee_ru", TestCase.first, TestCase.second.first,
                *Result, TestCase.second.second);
  }
}

__global__ void fmaf_ieee_rz(float *const Result, float Input1, float Input2,
                             float Input3) {
  *Result = __fmaf_ieee_rz(Input1, Input2, Input3);
}

void testFmaf_ieee_rzCases(
    const vector<pair<vector<float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmaf_ieee_rz<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                           TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fmaf_ieee_rz", TestCase.first, TestCase.second.first,
                *Result, TestCase.second.second);
  }
}

__global__ void fmul_rd(float *const Result, float Input1, float Input2) {
  *Result = __fmul_rd(Input1, Input2);
}

void testFmul_rdCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmul_rd<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fmul_rd", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fmul_rn(float *const Result, float Input1, float Input2) {
  *Result = __fmul_rn(Input1, Input2);
}

void testFmul_rnCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmul_rn<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fmul_rn", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fmul_ru(float *const Result, float Input1, float Input2) {
  *Result = __fmul_ru(Input1, Input2);
}

void testFmul_ruCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmul_ru<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fmul_ru", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fmul_rz(float *const Result, float Input1, float Input2) {
  *Result = __fmul_rz(Input1, Input2);
}

void testFmul_rzCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fmul_rz<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fmul_rz", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fsub_rd(float *const Result, float Input1, float Input2) {
  *Result = __fsub_rd(Input1, Input2);
}

void testFsub_rdCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fsub_rd<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fsub_rd", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fsub_rn(float *const Result, float Input1, float Input2) {
  *Result = __fsub_rn(Input1, Input2);
}

void testFsub_rnCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fsub_rn<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fsub_rn", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fsub_ru(float *const Result, float Input1, float Input2) {
  *Result = __fsub_ru(Input1, Input2);
}

void testFsub_ruCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fsub_ru<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fsub_ru", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fsub_rz(float *const Result, float Input1, float Input2) {
  *Result = __fsub_rz(Input1, Input2);
}

void testFsub_rzCases(
    const vector<pair<pair<float, float>, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fsub_rz<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__fsub_rz", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

int main() {
  testExpfCases({
      {-0.3, {0.7408, 4}},
      {0.34, {1.405, 3}},
      {23, {9745000000, -6}},
      {-12, {0.000006144, 9}},
  });
  testNorm3dfCases({
      {{-0.3, -0.34, -0.98}, {1.079814791679382, 15}},
      {{0.3, 0.34, 0.98}, {1.079814791679382, 15}},
      {{0.5, 456, 23}, {456.5799560546875, 13}},
      {{23, 432, 23}, {433.2228088378906, 13}},
  });
  testNorm4dfCases({
      {{-0.3, -0.34, -0.98, 1}, {1.471734, 6}},
      {{0.3, 0.34, 0.98, 1}, {1.471734, 6}},
      {{0.5, 456, 23, 1}, {456.5810546875, 13}},
      {{23, 432, 23, 1}, {433.2239685058594, 13}},
  });
  testNormcdffCases({
      {-5, {0.0000002866515842470108, 22}},
      {-3, {0.001349898055195808, 18}},
      {0, {0.5, 16}},
      {1, {0.8413447, 7}},
      {5, {0.9999997019767761, 16}},
  });
  testNormfCases({
      {{-0.3, -0.34, -0.98}, {1.079814791679382, 15}},
      {{0.3, 0.34, 0.98}, {1.079814791679382, 15}},
      {{0.5}, {0.5, 16}},
      {{23, 432, 23, 456, 23}, {629.402099609375, 13}},
  });
  testRcbrtfCases({
      {-0.3, {-1.494, 3}},
      {0.3, {1.494, 3}},
      {0.5, {1.26, 3}},
      {23, {0.3516, 4}},
  });
  testRnorm3dfCases({
      {{-0.3, -0.34, -0.98}, {0.9261, 4}},
      {{0.3, 0.34, 0.98}, {0.9261, 4}},
      {{0.5, 456, 23}, {0.0021902, 7}},
      {{23, 432, 23}, {0.0023083, 7}},
  });
  testRnorm4dfCases({
      {{-0.3, -0.34, -0.98, 1}, {0.6795, 4}},
      {{0.3, 0.34, 0.98, 1}, {0.6795, 4}},
      {{0.5, 456, 23, 1}, {0.0021902, 7}},
      {{23, 432, 23, 1}, {0.0023083, 7}},
  });
  testRnormfCases({
      {{-0.3, -0.34, -0.98}, {0.9261, 4}},
      {{0.3, 0.34, 0.98}, {0.9261, 4}},
      {{0.5}, {2, 3}},
      {{23, 432, 23, 456, 23}, {0.0015888, 7}},
  });
  test_ExpfCases({
      {-0.3, {0.7408, 4}},
      {0.34, {1.405, 3}},
      {23, {9745000000, -6}},
      {-12, {0.000006144, 9}},
  });
  testFadd_rdCases({
      {{-0.3, -0.4}, {-0.7000000476837158, 16}},
      {{0.3, -0.4}, {-0.09999999403953552, 17}},
      {{0.3, 0.4}, {0.7, 7}},
      {{0.3, 0.8}, {1.100000023841858, 15}},
      {{3, 4}, {7, 15}},
  });
  testFadd_rnCases({
      {{-0.3, -0.4}, {-0.7000000476837158, 16}},
      {{0.3, -0.4}, {-0.09999999403953552, 17}},
      {{0.3, 0.4}, {0.7000000476837158, 16}},
      {{0.3, 0.8}, {1.100000023841858, 15}},
      {{3, 4}, {7, 15}},
  });
  testFadd_ruCases({
      {{-0.3, -0.4}, {-0.7, 7}},
      {{0.3, -0.4}, {-0.09999999403953552, 17}},
      {{0.3, 0.4}, {0.7000000476837158, 16}},
      {{0.3, 0.8}, {1.100000023841858, 15}},
      {{3, 4}, {7, 15}},
  });
  testFadd_rzCases({
      {{-0.3, -0.4}, {-0.7, 7}},
      {{0.3, -0.4}, {-0.09999999403953552, 17}},
      {{0.3, 0.4}, {0.7, 7}},
      {{0.3, 0.8}, {1.100000023841858, 15}},
      {{3, 4}, {7, 15}},
  });
  testFmaf_rdCases({
      {{-0.3, -0.4, -0.2}, {-0.07999999821186066, 17}},
      {{0.3, -0.4, -0.1}, {-0.2200000137090683, 16}},
      {{0.3, 0.4, 0.1}, {0.22, 7}},
      {{0.3, 0.4, 0}, {0.12000000476837158, 17}},
      {{3, 4, 5}, {17, 14}},
  });
  testFmaf_rnCases({
      {{-0.3, -0.4, -0.2}, {-0.07999999821186066, 17}},
      {{0.3, -0.4, -0.1}, {-0.2200000137090683, 16}},
      {{0.3, 0.4, 0.1}, {0.2200000137090683, 16}},
      {{0.3, 0.4, 0}, {0.12000000476837158, 17}},
      {{3, 4, 5}, {17, 14}},
  });
  testFmaf_ruCases({
      {{-0.3, -0.4, -0.2}, {-0.07999999, 8}},
      {{0.3, -0.4, -0.1}, {-0.22, 7}},
      {{0.3, 0.4, 0.1}, {0.2200000137090683, 16}},
      {{0.3, 0.4, 0}, {0.12000001, 8}},
      {{3, 4, 5}, {17, 14}},
  });
  testFmaf_rzCases({
      {{-0.3, -0.4, -0.2}, {-0.07999999, 8}},
      {{0.3, -0.4, -0.1}, {-0.22, 7}},
      {{0.3, 0.4, 0.1}, {0.22, 7}},
      {{0.3, 0.4, 0}, {0.12000000476837158, 17}},
      {{3, 4, 5}, {17, 14}},
  });

  testFmaf_ieee_rdCases({
      {{-0.3, -0.4, -0.2}, {-0.07999999821186066, 17}},
      {{0.3, -0.4, -0.1}, {-0.2200000137090683, 16}},
      {{0.3, 0.4, 0.1}, {0.22, 7}},
      {{0.3, 0.4, 0}, {0.12000000476837158, 17}},
      {{3, 4, 5}, {17, 14}},
  });
  testFmaf_ieee_rnCases({
      {{-0.3, -0.4, -0.2}, {-0.07999999821186066, 17}},
      {{0.3, -0.4, -0.1}, {-0.2200000137090683, 16}},
      {{0.3, 0.4, 0.1}, {0.2200000137090683, 16}},
      {{0.3, 0.4, 0}, {0.12000000476837158, 17}},
      {{3, 4, 5}, {17, 14}},
  });
  testFmaf_ieee_ruCases({
      {{-0.3, -0.4, -0.2}, {-0.07999999, 8}},
      {{0.3, -0.4, -0.1}, {-0.22, 7}},
      {{0.3, 0.4, 0.1}, {0.2200000137090683, 16}},
      {{0.3, 0.4, 0}, {0.12000001, 8}},
      {{3, 4, 5}, {17, 14}},
  });
  testFmaf_ieee_rzCases({
      {{-0.3, -0.4, -0.2}, {-0.07999999, 8}},
      {{0.3, -0.4, -0.1}, {-0.22, 7}},
      {{0.3, 0.4, 0.1}, {0.22, 7}},
      {{0.3, 0.4, 0}, {0.12000000476837158, 17}},
      {{3, 4, 5}, {17, 14}},
  });

  testFmul_rdCases({
      {{-0.3, -0.4}, {0.12000000476837158, 17}},
      {{0.3, -0.4}, {-0.12000001, 8}},
      {{0.3, 0.4}, {0.12000000476837158, 17}},
      {{0.3, 0.8}, {0.2400000095367432, 16}},
      {{3, 4}, {12, 15}},
  });
  testFmul_rnCases({
      {{-0.3, -0.4}, {0.12000000476837158, 17}},
      {{0.3, -0.4}, {-0.12000000476837158, 17}},
      {{0.3, 0.4}, {0.12000000476837158, 17}},
      {{0.3, 0.8}, {0.2400000095367432, 16}},
      {{3, 4}, {12, 15}},
  });
  testFmul_ruCases({
      {{-0.3, -0.4}, {0.12000001, 8}},
      {{0.3, -0.4}, {-0.12000000476837158, 17}},
      {{0.3, 0.4}, {0.12000001, 8}},
      {{0.3, 0.8}, {0.24, 7}},
      {{3, 4}, {12, 15}},
  });
  testFmul_rzCases({
      {{-0.3, -0.4}, {0.12000000476837158, 17}},
      {{0.3, -0.4}, {-0.12000000476837158, 17}},
      {{0.3, 0.4}, {0.12000000476837158, 17}},
      {{0.3, 0.8}, {0.2400000095367432, 16}},
      {{3, 4}, {12, 15}},
  });
  testFsub_rdCases({
      {{-0.3, -0.4}, {0.09999999403953552, 17}},
      {{0.3, -0.4}, {0.7, 7}},
      {{0.3, 0.4}, {-0.09999999403953552, 17}},
      {{0.3, 0.8}, {-0.5, 16}},
      {{3, 4}, {-1, 15}},
  });
  testFsub_rnCases({
      {{-0.3, -0.4}, {0.09999999403953552, 17}},
      {{0.3, -0.4}, {0.7000000476837158, 16}},
      {{0.3, 0.4}, {-0.09999999403953552, 17}},
      {{0.3, 0.8}, {-0.5, 16}},
      {{3, 4}, {-1, 15}},
  });
  testFsub_ruCases({
      {{-0.3, -0.4}, {0.09999999403953552, 17}},
      {{0.3, -0.4}, {0.7000000476837158, 16}},
      {{0.3, 0.4}, {-0.09999999403953552, 17}},
      {{0.3, 0.8}, {-0.5, 16}},
      {{3, 4}, {-1, 15}},
  });
  testFsub_rzCases({
      {{-0.3, -0.4}, {0.09999999403953552, 17}},
      {{0.3, -0.4}, {0.7, 7}},
      {{0.3, 0.4}, {-0.09999999403953552, 17}},
      {{0.3, 0.8}, {-0.5, 16}},
      {{3, 4}, {-1, 15}},
  });
  cout << "passed " << passed << "/" << passed + failed << " cases!" << endl;
  if (failed) {
    cout << "failed!" << endl;
  }
  return failed;
}
