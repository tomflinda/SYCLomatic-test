// ===------- graph.cu ------------------------------------ *- CUDA -* ---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>

const int blockSize = 256;
const int numBlocks = (10 + blockSize - 1) / blockSize;

__global__ void init(float *a) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < 10) {
    a[id] = 1.0f;
  }
}

__global__ void incrementA(float *a) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < 10) {
    a[id] += 1.0f;
  }
}

int main() {

  hipGraph_t graph;

  hipStream_t stream;

  hipStreamCreate(&stream);

  float *d_a, h_a[10];

  hipMalloc(&d_a, 10 * sizeof(float));

  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

  init<<<numBlocks, blockSize, 0, stream>>>(d_a);

  incrementA<<<numBlocks, blockSize, 0, stream>>>(d_a);

  hipStreamEndCapture(stream, &graph);
  hipGraphExec_t execGraph;
  hipGraphInstantiate(&execGraph, graph, NULL, NULL, 0);

  hipGraphLaunch(execGraph, stream);

  hipStreamSynchronize(stream); // Ensure the graph has completed execution

  hipMemcpy(h_a, d_a, 10 * sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < 10; i++) {
    if (h_a[i] != 2.0f) {
      printf("Results do not match\n");
      return -1;
    }
  }

  size_t numNodes;
  hipGraphGetNodes(graph, nullptr, &numNodes);
  std::vector<hipGraphNode_t> nodes(numNodes);
  hipGraphGetNodes(graph, nodes.data(), &numNodes);

  // Get root nodes in the graph
  size_t numRootNodes;
  hipGraphGetRootNodes(graph, nullptr, &numRootNodes);
  std::vector<hipGraphNode_t> rootNodes(numRootNodes);
  hipGraphGetRootNodes(graph, rootNodes.data(), &numRootNodes);

  if (numNodes != 2 || numRootNodes != 1) {
    printf("Number of nodes or root nodes do not match\n");
    return -1;
  }

  printf("Passed\n");

  hipStreamDestroy(stream);
  hipFree(d_a);
  hipGraphExecDestroy(execGraph);
  hipGraphDestroy(graph);

  return 0;
}
