#include<hip/hip_runtime.h>
#include<iostream>

void hostCallback(void *userData) {
  const char *msg = static_cast<const char*>(userData);
  std::cout << "Host callback executed. Message: " << msg << std::endl;
}

int main() {
  hipStream_t stream;
  hipStreamCreate(&stream);

  hipError_t err;
  const char *message = "Kernel execution finished.";

  err = hipLaunchHostFunc(stream, hostCallback, (void*)message);

  hipHostFn_t fn = hostCallback;
  hipLaunchHostFunc(stream, fn, (void*)message);

  hipStreamDestroy(stream);
  return 0;
}

