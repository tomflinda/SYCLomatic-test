#include "hip/hip_runtime.h"
// ====------ cooperative_groups.cu --------------------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_cooperative_groups.h>
#include <cstdio>
namespace cg = cooperative_groups;

// threadIdx.x: 0 ... 7, 8 ... 15, 16 ... 23, 24 ... 31, 32 ... 39, 40 ... 47, 48 ... 55
//              -------  --------  ---------  ---------  ---------  ---------  ---------
//              0        1         2          3          4          5          6

bool verify_array(unsigned int *expected, unsigned int *res, unsigned int size) {
  for (unsigned int i = 0; i < size; i++) {
    if (expected[i] != res[i]) {
      return false;
    }
  }
  return true;
}

__global__ void kernel(unsigned int *data, unsigned int *result) {
  cg::thread_block ttb = cg::this_thread_block();
  cg::thread_block_tile<8> tbt8 = cg::tiled_partition<8>(ttb);
  auto thread_this = cg::this_thread();
  unsigned int temp = data[threadIdx.x];
  temp = tbt8.shfl_down(temp, 1);
  data[threadIdx.x] = temp;

  if (threadIdx.x == 50) {
    result[0] = tbt8.size();
    result[1] = tbt8.thread_rank();
    result[2] = ttb.size();
  }
}

__global__ void test_group_thread_index(unsigned int *data) {
  cg::thread_block ttb = cg::this_thread_block();
  auto group_x = ttb.group_index().x;
  auto thread_x = ttb.thread_index().x;
  data[threadIdx.x] = group_x + thread_x;
}

int main() {
  unsigned int result_host[3];
  unsigned int data_host[56];
  unsigned int data_ret[56];
  result_host[2] = 0;
  for (int i = 0; i < 56; i++) {
    data_host[i] = i;
  }

  unsigned int *result_device, *data_device;
  hipMalloc(&result_device, sizeof(unsigned int) * 3);
  hipMalloc(&data_device, sizeof(unsigned int) * 56);

  hipMemcpy(data_device, &data_host, sizeof(unsigned int) * 56, hipMemcpyHostToDevice);
  kernel<<<1, 56>>>(data_device, result_device);
  hipMemcpy(result_host, result_device, sizeof(unsigned int) * 3, hipMemcpyDeviceToHost);
  hipMemcpy(&data_host, data_device, sizeof(unsigned int) * 56, hipMemcpyDeviceToHost);
  hipFree(result_device);
  hipFree(data_device);

  bool checker1 = false;
  unsigned int expected[56] = {
    1, 2, 3, 4, 5, 6, 7, 7,
    9, 10, 11, 12, 13, 14, 15, 15,
    17, 18, 19, 20, 21, 22, 23, 23,
    25, 26, 27, 28, 29, 30, 31, 31,
    33, 34, 35, 36, 37, 38, 39, 39,
    41, 42, 43, 44, 45, 46, 47, 47,
    49, 50, 51, 52, 53, 54, 55, 55
  };
  if (verify_array(expected, data_host, 56)) {
    checker1 = true;
  } else {
    printf("checker1 failed\n");
    for (int i = 0; i < 7; i++) {
      for (int j = 0; j < 8; j++) {
        int idx = i * 8 + j;
        printf("%d, ", data_host[idx]);
      }
      printf("\n");
    }
  }

  bool checker2 = false;
  if (result_host[0] == 8 &&
      result_host[1] == 2 &&
      result_host[2] != 0) {
    checker2 = true;
  } else {
    printf("checker2 failed\n");
    printf("%d, %d\n", result_host[0], result_host[1]);
  }

  unsigned int *result_device_kernel_2;
  hipMalloc(&result_device_kernel_2, sizeof(unsigned int) * 56);
  test_group_thread_index<<<2, 56>>>(result_device_kernel_2);
  hipMemcpy(data_ret, result_device_kernel_2, sizeof(unsigned int) * 56, hipMemcpyDeviceToHost);
  unsigned int expected_ret[56] = {
    1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15,
    16, 17, 18, 19, 20, 21, 22, 23, 24, 25, 26, 27, 28,
    29, 30, 31, 32, 33, 34, 35, 36, 37, 38, 39, 40, 41,
    42, 43, 44, 45, 46, 47, 48, 49, 50,
    51, 52, 53, 54, 55, 56
  };
  bool checker3 = false;

  if (verify_array(expected_ret, data_ret, 56)) {
    checker3 = true;
  } else {
    printf("checker3 failed\n");
  }


  if (checker1 && checker2 && checker3)
    return 0;
  return -1;
}
