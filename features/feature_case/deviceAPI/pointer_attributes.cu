// ====------ pointer_attributes.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <iostream>
int main() {
  int N = 2048;
  size_t size = N * sizeof(float);

  float *h_A = (float *)malloc(size);
  float *h_B = (float *)malloc(size);

  float *d_A;
  float *d_B;
  float *d_C;
  hipMalloc((void **)&d_A, size);
  hipMalloc((void **)&d_B, size);
  hipMalloc((void **)&d_C, size);

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  void * malloc_host;
  hipHostMalloc((void **)&malloc_host, size, hipHostMallocDefault);
  hipPointerAttribute_t attributes2;
  hipPointerGetAttributes (&attributes2, malloc_host);
  std::cout << "====== Malloc Host Attributes =======" << std::endl;
  std::cout << "malloc host " << malloc_host << std::endl;
  std::cout << attributes2.device << std::endl;
  std::cout << attributes2.hostPointer << std::endl;
  std::cout << attributes2.devicePointer << std::endl;

  hipPointerAttribute_t *attributes3 = new hipPointerAttribute_t();
  hipPointerGetAttributes (attributes3, d_A);
  std::cout << "====== Device Attributes =======" << std::endl;
  std::cout << attributes3->device << std::endl;
  std::cout << attributes3->hostPointer << std::endl;
  std::cout << attributes3->devicePointer << std::endl;
  if (attributes3->type == hipMemoryTypeHost) {
    return 0;
  } else if (attributes3->type == hipMemoryTypeDevice) {
    return 0;
  } else if (attributes3->type == hipMemoryTypeManaged) {
    return 0;
  } else if (attributes3->type == cudaMemoryTypeUnregistered) {
    return 0;
  }
}
