// ====------ driverMem.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <vector>
#include <algorithm>

void test1(){
    size_t result1, result2;
    int size = 32;
    float* f_A;
    hipDeviceptr_t f_D = 0;
    hipDeviceptr_t f_D2 = 0;
    hipError_t r;

    hipHostAlloc((void **)&f_A, size, hipHostMallocMapped);

    hipMemAllocHost((void **)&f_A, size);

    hipMalloc(&f_D, size);

    hipMallocManaged(&f_D, size, hipMemAttachHost);


    hipStream_t stream;

    hipMemcpyHtoDAsync(f_D, f_A, size, stream);

    hipMemcpyHtoDAsync(f_D, f_A, size, 0);

    hipMemcpyHtoD(f_D, f_A, size);

    hipMemcpyDtoDAsync(f_D, f_D2, size, stream);

    r = hipMemcpyDtoDAsync(f_D, f_D2, size, stream);

    hipMemcpyDtoDAsync(f_D, f_D2, size, 0);

    r = hipMemcpyDtoDAsync(f_D, f_D2, size, 0);


    hipMemcpyDtoD(f_D, f_D2, size);

    r = hipMemcpyDtoD(f_D, f_D2, size);

    hipMemcpyDtoHAsync(f_A, f_D, size, stream);

    hipMemcpyDtoHAsync(f_A, f_D, size, 0);

    hipMemcpyDtoH(f_A, f_D, size);

    cuMemcpy(f_D, f_D2, size);
    r = cuMemcpy(f_D, f_D2, size);

    cuMemcpyAsync(f_D, f_D2, size, stream);
    r = cuMemcpyAsync(f_D, f_D2, size, stream);

    cuMemcpyAsync(f_D, f_D2, size, 0);
    r = cuMemcpyAsync(f_D, f_D2, size, 0);


    hipHostGetDevicePointer(&f_D, f_A, 0);

    hip_Memcpy2D cpy;

    cpy.dstMemoryType = hipMemoryTypeHost;

    cpy.dstHost = f_A;

    cpy.dstPitch = 20;

    cpy.dstY = 10;

    cpy.dstXInBytes = 15;


    cpy.srcMemoryType = hipMemoryTypeDevice;

    cpy.srcDevice = f_D;

    cpy.srcPitch = 20;

    cpy.srcY = 10;

    cpy.srcXInBytes = 15;


    cpy.WidthInBytes = 4;

    cpy.Height = 7;


    hipMemcpyParam2D(&cpy);

    hipDrvMemcpy2DUnaligned(&cpy);

    hipMemcpyParam2DAsync(&cpy, stream);

    HIP_MEMCPY3D cpy2;

    hipArray_t ca;

    cpy2.dstMemoryType = hipMemoryTypeArray;

    cpy2.dstArray = ca;

    cpy2.dstPitch = 5;

    cpy2.dstHeight = 4;

    cpy2.dstY = 3;

    cpy2.dstZ = 2;

    cpy2.dstXInBytes = 1;

    cpy2.dstLOD = 0;


    cpy2.srcMemoryType = hipMemoryTypeHost;

    cpy2.srcHost = f_A;

    cpy2.srcPitch = 5;

    cpy2.srcHeight = 4;

    cpy2.srcY = 3;

    cpy2.srcZ = 2;

    cpy2.srcXInBytes = 1;

    cpy2.srcLOD = 0;


    cpy2.WidthInBytes = 3;

    cpy2.Height = 2;

    cpy2.Depth = 1;

    hipDrvMemcpy3D(&cpy2);

    float *h_A = (float *)malloc(100);
    hipHostFree(h_A);

    unsigned int* pFlags;

    hipMemAllocHost((void **)&f_A, size);

    hipHostGetFlags(pFlags, f_A);

    hipHostRegister((void *)pFlags, size, hipHostRegisterPortable);

    hipHostUnregister((void *)pFlags);
}

int test2() {
  int ret = 0;
  constexpr int size = 64;
  int v1[size];
  int v2[size];

  hipDeviceptr_t p1 = (hipDeviceptr_t)v1;
  hipDeviceptr_t p2 = (hipDeviceptr_t)v2;
  hipDeviceptr_t q1;
  hipDeviceptr_t q2;

  // check if v1 and v2 agree on first i elements

  auto check = [&](int i, std::string fail) {
    if (!std::equal(v1, v1+i, v2)) {
      std::cout << fail << "\n";
      ret = 1;
    }
  };

  // v1 = {0, 1, 2, ...}
  // v2 = {-1, -1, ...}
  auto initialize = [&]() {
    for (int i = 0; i < size; ++i) {
      v1[i] = i;
      v2[i] = -1;
    }
    hipMalloc(&q1, sizeof(int)*size);
    hipMalloc(&q2, sizeof(int)*size);
  };

  for (int i = 1; i < size; i *= 2) {
    int n = sizeof(int)*i;

    // host to host copy
    initialize();
    cuMemcpy(p2, p1, n);
    check(i, "cuMemcpy fail " + std::to_string(i));

    // host to device copy async, device to host copy
    initialize();
    cuMemcpyAsync(q1, p1, n, 0);
    hipStreamSynchronize(0);
    cuMemcpy(p2, q1, n);
    check(i, "cuMemcpyAsync 1 fail " + std::to_string(i));

    // host to device copy, device to device async copy,
    // device to host copy
    initialize();
    cuMemcpy(q1, p1, n);
    cuMemcpyAsync(q2, q1, n, 0);
    hipStreamSynchronize(0);
    cuMemcpy(p2, q2, n);
    check(i, "cuMemcpyAsync 2 fail " + std::to_string(i));
  }

  return ret;
}

int main() {
  hipInit(0);
  hipDevice_t dev = 0;
  hipDeviceGet(&dev, 0);
  hipCtx_t ctx = 0;
  hipCtxCreate(&ctx, 0, dev);
  return test2();
}
