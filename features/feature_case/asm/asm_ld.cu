// ====------ asm_ld.cu ----------------------------------- *- CUDA -* ---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===---------------------------------------------------------------------===//

#include <hip/hip_runtime.h>

#define TEST(FN)                                                               \
  {                                                                            \
    if (FN()) {                                                                \
      printf("Test " #FN " PASS\n");                                           \
    } else {                                                                   \
      printf("Test " #FN " FAIL\n");                                           \
      return 1;                                                                \
    }                                                                          \
  }

__device__ inline void load_global_short4(short4 &a, const short4 *addr) {
  short x, y, z, w;
  asm("ld.cg.global.v4.s16 {%0, %1, %2, %3}, [%4+0];"
      : "=h"(x), "=h"(y), "=h"(z), "=h"(w)
      : "l"(addr));
  a.x = x;
  a.y = y;
  a.z = z;
  a.w = w;
}

__global__ void test_kernel(short4 *d_out, const short4 *d_in) {
  short4 val;
  load_global_short4(val, d_in);
  *d_out = val;
}

__device__ inline void load_global_short2(short2 &a, const short2 *addr) {
  short x, y, z, w;
  asm("ld.cg.global.v2.s16 {%0, %1}, [%2+0];" : "=h"(x), "=h"(y) : "l"(addr));
  a.x = x;
  a.y = y;
}

__global__ void test_kernel(short2 *d_out, const short2 *d_in) {
  short2 val;
  load_global_short2(val, d_in);
  *d_out = val;
}

bool test_1() {
  short4 h_in = {1, 2, 3, 4};
  short4 h_out;
  short4 *d_in, *d_out;

  hipMalloc(&d_in, sizeof(short4));
  hipMalloc(&d_out, sizeof(short4));
  hipMemcpy(d_in, &h_in, sizeof(short4), hipMemcpyHostToDevice);

  test_kernel<<<1, 1>>>(d_out, d_in);
  hipMemcpy(&h_out, d_out, sizeof(short4), hipMemcpyDeviceToHost);

  hipFree(d_in);
  hipFree(d_out);

  return (h_out.x == h_in.x && h_out.y == h_in.y && h_out.z == h_in.z &&
          h_out.w == h_in.w)
             ? true
             : false;
}

bool test_2() {
  short2 h_in = {1, 2};
  short2 h_out;
  short2 *d_in, *d_out;

  hipMalloc(&d_in, sizeof(short2));
  hipMalloc(&d_out, sizeof(short2));
  hipMemcpy(d_in, &h_in, sizeof(short2), hipMemcpyHostToDevice);

  test_kernel<<<1, 1>>>(d_out, d_in);
  hipMemcpy(&h_out, d_out, sizeof(short2), hipMemcpyDeviceToHost);

  hipFree(d_in);
  hipFree(d_out);

  return (h_out.x == h_in.x && h_out.y == h_in.y) ? true : false;
}

__device__ __forceinline__ int ld_flag_volatile(int *flag_addr) {
  int flag;
  asm volatile("ld.volatile.global.u32 %0, [%1]; membar.gl;"
               : "=r"(flag)
               : "l"(flag_addr));
  return flag;
}

__global__ void test_ld_flag_acquire(int *flag_addr, int *out_value) {
  int val = ld_flag_volatile(flag_addr);
  *out_value = val;
}

bool test_3() {

  int h_flag_value = 999;
  int h_result = 0;

  int *d_flag_addr;
  int *d_result;

  hipMalloc(&d_flag_addr, sizeof(int));
  hipMalloc(&d_result, sizeof(int));

  hipMemcpy(d_flag_addr, &h_flag_value, sizeof(int), hipMemcpyHostToDevice);
  hipMemset(d_result, 0, sizeof(int));

  test_ld_flag_acquire<<<1, 1>>>(d_flag_addr, d_result);
  hipDeviceSynchronize();

  hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

  hipFree(d_flag_addr);
  hipFree(d_result);

  return (h_result == h_flag_value) ? true : false;
}

int main() {
  TEST(test_1);
  TEST(test_2);
  TEST(test_3);
  return 0;
}