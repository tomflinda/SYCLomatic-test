#include "hip/hip_runtime.h"
// ===--------------- nvshmem.cu --------------- *- CUDA -* ---------------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===---------------------------------------------------------------------===//

#include <nvshmem.h>
#include <nvshmemx.h>
#include <mpi.h>
#include <iostream>
#include <cassert>

#define N 32

__global__ void set_data(int *shared_data, int mype) {
  size_t i = threadIdx.x;

  shared_data[i] = static_cast<int>(mype * 2);
}

__global__ void kernel_putmem_signal_nbi(int *shared_data, uint64_t *signal, int val, int target_pe) {
  nvshmem_putmem_signal_nbi(shared_data, shared_data, N * sizeof(int), signal, val, NVSHMEM_SIGNAL_SET, target_pe);
}

__global__ void kernel_signal_wait_until(uint64_t *signal, uint64_t val) {
  nvshmem_signal_wait_until(signal, NVSHMEM_CMP_EQ, val);
}

int main(int argc, char **argv) {
  MPI_Init(&argc, &argv);
  int rank, nranks;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &nranks);

  nvshmemx_init_attr_t attr;
  MPI_Comm mpi_comm = MPI_COMM_WORLD;

  attr.mpi_comm = &mpi_comm;
  nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

  int mype = nvshmem_my_pe();
  int npes = nvshmem_n_pes();

  std::cout << "ISHMEM initialized with " << npes << " PEs." << std::endl;
  std::cout << "My PE: " << mype << std::endl;

  int *shared_data = (int *)nvshmem_malloc(N * sizeof(int));
  assert(shared_data != nullptr && "nvshmem_malloc failed");

  uint64_t *signal_addr = (uint64_t *)nvshmem_malloc(sizeof(uint64_t));
  assert(signal_addr != nullptr && "nvshmem_malloc for signal failed");

  set_data<<<N, N>>>(shared_data, mype);

  const int target_pe = 1;

  // copy data from PE 0 to PE 1
  if (mype == 0) {
    nvshmem_putmem_nbi(shared_data, shared_data, N * sizeof(int), target_pe);
  }

  int recv_shared_data[N] = {1};

  // Retrieve the data on PE 1
  if (mype == 1) {
    hipMemcpy((void *)recv_shared_data, (void *)shared_data, N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
      if (recv_shared_data[i] != 0) {
        std::cerr << "[" << mype << "] Data verification 1 failed at index " << i << ": " << recv_shared_data[i] << "\n";
        std::exit(1);
      }
    }
  }

  // Get the pointer to data on PE 1
  int *remote_data = (int *)nvshmem_ptr(shared_data, 1);
  hipMemcpy((void *)recv_shared_data, (void *)remote_data, N * sizeof(int), hipMemcpyDeviceToHost);

  if (mype == 0) {
    for (int i = 0; i < N; i++) {
      if (recv_shared_data[i] != 0) {
        std::cerr << "[" << mype << "] Data verification 2 failed at index " << i << ": " << recv_shared_data[i] << "\n";
        std::exit(1);
      }
    }
  }
  std::cout << "putmem_nbi & shmem_ptr: Data verification successful" << std::endl;

  // Reset data on all PEs
  set_data<<<N, N>>>(shared_data, mype);

  // Allocate & set signal memory
  uint64_t *signal = (uint64_t *)nvshmem_malloc(sizeof(uint64_t));
  assert(signal != nullptr && "nvshmem_malloc failed");
  
  uint64_t h_signal = 0;
  hipMemcpy((void *)signal, (void *)&h_signal, sizeof(uint64_t), hipMemcpyHostToDevice);

  // Copy data from PE 0 to PE 1 and signal completion
  // nvshmem_barrier_all();
  kernel_putmem_signal_nbi<<<1, 1>>>(shared_data, signal, 1, 1);

  // Check whether signal value & data updated in PE 1
  if (mype == 1) {
    kernel_signal_wait_until<<<1, 1>>>(signal, 1);

    hipMemcpy((void *)recv_shared_data, (void *)shared_data, N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
      if (recv_shared_data[i] != 0) {
        std::cerr << "[" << mype << "] Data verification 3 failed at index " << i << ": " << recv_shared_data[i] << "\n";
        std::exit(1);
      }
    }
  }
  std::cout << "putmem_signal_nbi & signal_wait_until: Data verification successful" << std::endl;

  // Set and update signal value in PE 1
  nvshmemx_signal_op(signal, 1, NVSHMEM_SIGNAL_SET, 1);
  nvshmemx_signal_op(signal, 1, NVSHMEM_SIGNAL_ADD, 1);

  // Check whether signal value updated in PE 1
  if (mype == 1) {
    kernel_signal_wait_until<<<1, 1>>>(signal, 2);

    hipMemcpy((void *)recv_shared_data, (void *)signal, sizeof(uint64_t), hipMemcpyDeviceToHost);

    if (recv_shared_data[0] != 2) {
      std::cerr << "[" << mype << "] Signal verification failed: " << recv_shared_data[0] << "\n";
      std::exit(1);
    }
  }
  std::cout << "signal_set/add: Data verification successful" << std::endl;

  nvshmem_free(shared_data);
  nvshmem_free(signal);

  nvshmem_finalize();
  MPI_Finalize();

  return 0;
}
