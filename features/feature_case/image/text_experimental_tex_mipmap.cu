#include "hip/hip_runtime.h"
// ===------------- text_experimental_tex_mipmap.cu ------ *- CUDA -* ------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===---------------------------------------------------------------------===//
#include <hip/hip_runtime.h>
#include <iostream>

const int height = 2;
const int width  = 4;
const int depth  = 2;

void set_3D_descriptor(HIP_ARRAY3D_DESCRIPTOR &desc) {
  desc.Width = width;
  desc.Depth = depth;
  desc.Height = height;
  desc.Format = HIP_AD_FORMAT_SIGNED_INT16;
  desc.NumChannels = 4;
}

int main() {
  hipDevice_t device;
  hipCtx_t context;

  // Initialize CUDA
  hipError_t result = hipInit(0);
  if (result != hipSuccess) {
    std::cerr << "Failed to initialize CUDA\n";
    return -1;
  }

  // Get the device
  result = hipDeviceGet(&device, 0);
  if (result != hipSuccess) {
    std::cerr << "Failed to get CUDA device\n";
    return -1;
  }

  // Create a context
  result = hipCtxCreate(&context, 0, device);
  if (result != hipSuccess) {
    std::cerr << "Failed to create CUDA context\n";
    return -1;
  }

  HIP_ARRAY3D_DESCRIPTOR desc;
  set_3D_descriptor(desc);

  hipMipmappedArray_t mmArray;
  unsigned int numMipmapLevels = 2;
  result = hipMipmappedArrayCreate(&mmArray, &desc, numMipmapLevels);
  if (result != hipSuccess) {
    std::cerr << "Failed to create mipmapped array\n";
    return -1;
  }

  hipArray_t level_arr;
  result = hipMipmappedArrayGetLevel(&level_arr, mmArray, 0);  // Get level 0
  if (result != hipSuccess) {
    std::cerr << "Failed to get mipmap level\n";
    return -1;
  }
  
  short4 mm1[height * width * depth] = {
    {1,  2, 3, 4},   {5, 6, 7, 8},   {9, 10, 11, 12},   {13, 14, 15, 16},
    {17, 18, 19, 20},  {21, 22, 23, 24}, {25, 26, 27, 28}, {29, 30, 31, 32},

    {33, 34, 35, 36}, {37, 38, 39, 40}, {41, 42, 43, 44}, {45, 46, 47, 48},
    {49, 50, 51, 52}, {53, 54, 55, 56}, {57, 58, 59, 60}, {61, 62, 63, 64}
  };

  HIP_MEMCPY3D copyAssist{0};
  // specify source details
  copyAssist.srcHost = mm1;
  copyAssist.srcMemoryType = hipMemoryTypeHost;
  copyAssist.srcHeight = height;
  copyAssist.srcPitch = sizeof(short4) * width;

  // specify destination details
  copyAssist.dstArray = level_arr;
  copyAssist.dstMemoryType = hipMemoryTypeArray;

  // specify copy dimensions
  copyAssist.WidthInBytes = sizeof(short4) * width;
  copyAssist.Height = height;
  copyAssist.Depth = depth;

  result = hipDrvMemcpy3D(&copyAssist);
  if (result != hipSuccess) {
    std::cerr << "Copy from host to device failed for mipmaped array\n";
    return -1;
  }

  hipTexRef texRef{0};
  cuTexRefCreate(&texRef);
  hipTexRefSetMipmappedArray(texRef, mmArray, 0);

  hipTexRefSetMipmapFilterMode(texRef, HIP_TR_FILTER_MODE_POINT);

  HIPfilter_mode fm;
  hipTexRefGetMipmapFilterMode(&fm, texRef);
  if (fm != HIP_TR_FILTER_MODE_POINT) {
    std::cout << "Filter mode test failed";
    return -1;
  }

  float min_clamp, max_clamp;
  hipTexRefGetMipmapLevelClamp(&min_clamp, &max_clamp, texRef);

  hipMipmappedArray_t anotherArray;
  hipTexRefGetMipMappedArray(&anotherArray, texRef);
  hipMipmappedArrayDestroy(mmArray);

  return 0;
}
