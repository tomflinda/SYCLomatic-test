// ===------------- text_experimental_tex_mipmap.cu ------ *- CUDA -* ------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===---------------------------------------------------------------------===//
#include <hip/hip_runtime.h>
#include <iostream>

const int height = 2;
const int width  = 4;
const int depth  = 2;

void set_3D_descriptor(HIP_ARRAY3D_DESCRIPTOR &desc) {
  desc.Width = width;
  desc.Depth = depth;
  desc.Height = height;
  desc.Format = HIP_AD_FORMAT_SIGNED_INT16;
  desc.NumChannels = 2;
}

int main() {
  hipInit(0);

  HIP_ARRAY3D_DESCRIPTOR desc;

  set_3D_descriptor(desc);

  hipMipmappedArray_t mmArray;
  unsigned int numMipmapLevels = 2;
  hipMipmappedArrayCreate(&mmArray, &desc, numMipmapLevels);

  hipArray_t level_arr;
  hipMipmappedArrayGetLevel(&level_arr, mmArray, 1);
  
  short4 mm1[height * width * depth] = {
    {1,  2, 3, 4},   {5, 6, 7, 8},   {9, 10, 11, 12},   {13, 14, 15, 16},
    {17, 18, 19, 20},  {21, 22, 23, 24}, {25, 26, 27, 28}, {29, 30, 31, 32},

    {33, 34, 35, 36}, {37, 38, 39, 40}, {41, 42, 43, 44}, {45, 46, 47, 48},
    {49, 50, 51, 52}, {53, 54, 55, 56}, {57, 58, 59, 60}, {61, 62, 63, 64}
  };

  HIP_MEMCPY3D copyAssist{0};
  // specify source details
  copyAssist.srcHost = mm1;
  copyAssist.srcMemoryType = hipMemoryTypeHost;
  copyAssist.Height = height;
  copyAssist.Depth = depth;
  copyAssist.WidthInBytes = sizeof(short4) * width;
  copyAssist.srcPitch = sizeof(short4) * width;
  
  // specify destination details
  copyAssist.dstArray = level_arr;
  copyAssist.dstMemoryType = hipMemoryTypeArray;
  copyAssist.dstXInBytes = 0;
  copyAssist.dstY = 0;

  int testStatus = 0;

  hipError_t result = hipDrvMemcpy3D(&copyAssist);
  if (result != hipSuccess) {
    testStatus = -1;
    std::cout << "Copy from host to device failed for mipmaped array\n";
  }

  hipTexRef texRef{0};
  hipTexRefSetMipmappedArray(texRef, mmArray, 0);

  hipTexRefSetMipmapFilterMode(texRef, HIP_TR_FILTER_MODE_POINT);

  HIPfilter_mode fm;
  hipTexRefGetMipmapFilterMode(&fm, texRef);
  if (fm != HIP_TR_FILTER_MODE_POINT) {
    testStatus = -1;
    std::cout << "Filter mode test failed";
  }

  float min_clamp, max_clamp;
  hipTexRefGetMipmapLevelClamp(&min_clamp, &max_clamp, texRef);

  hipMipmappedArray_t anotherArray;
  hipTexRefGetMipMappedArray(&anotherArray, texRef);

  hipMipmappedArrayDestroy(mmArray);

  return testStatus;
}
