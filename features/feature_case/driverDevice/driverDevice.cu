// ====------ driverDevice.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define NUM 1
#define CUDA_SAFE_CALL( call) do {\
  int err = call;                \
} while (0)
int main(){
  int result1, result2;
  int *presult1 = &result1, *presult2 = &result2;
  hipDevice_t device, peerDevice;
  hipDevice_t *pdevice = &device;
  hipDeviceGet(&device, 0);
  hipDeviceGet(&device, NUM);
  hipDeviceGet(pdevice, 0);
  hipDeviceGet((hipDevice_t *)pdevice, 0);
  CUDA_SAFE_CALL(hipDeviceGet(&device, 0));
  hipDeviceGetAttribute(&result1, hipDeviceAttributeComputeCapabilityMajor, device);
  hipDeviceGetAttribute(&result1, hipDeviceAttributeComputeCapabilityMinor, device);
  hipDeviceGetAttribute(&result1, hipDeviceAttributeIntegrated, device);
  hipDeviceGetAttribute(&result1, hipDeviceAttributeClockRate, device);
  hipDeviceGetAttribute(&result1, hipDeviceAttributeMultiprocessorCount, device);
  hipDeviceGetAttribute(&result1, hipDeviceAttributeHostNativeAtomicSupported, device);
  CUDA_SAFE_CALL(hipDeviceGetAttribute(&result1, hipDeviceAttributeMultiprocessorCount, device));

  hipDeviceGetAttribute(&result1, hipDeviceAttributeTotalConstantMemory, device);
  hipDeviceGetAttribute(&result1, hipDeviceAttributeWarpSize, device);
  hipDeviceGetAttribute(&result1, hipDeviceAttributeMaxThreadsPerBlock, device);
  hipDeviceGetAttribute(&result1, hipDeviceAttributeTextureAlignment, device);
  hipDeviceGetAttribute(&result1, hipDeviceAttributeMaxRegistersPerBlock, device);

  hipDeviceComputeCapability(&result1, &result2, device);
  CUDA_SAFE_CALL(hipDeviceComputeCapability(&result1, &result2, device));
  CUDA_SAFE_CALL(hipGetDeviceCount(&result1));
  hipGetDeviceCount(&result1);
  CUDA_SAFE_CALL(hipGetDeviceCount(&result1));

  char name[100];
  hipDeviceGetName(name, 90, device);
  CUDA_SAFE_CALL(hipDeviceGetName(name, 90, device));

  hipDeviceGetAttribute(&result1, hipDeviceAttributeMaxBlockDimX, device);
  hipDeviceGetAttribute(&result1, hipDeviceAttributeMaxSharedMemoryPerBlock, device);

  hipDeviceP2PAttr p2p_attr = hipDevP2PAttrAccessSupported;
  hipDeviceGetP2PAttribute(&result1, p2p_attr, device, peerDevice);

  hipDeviceGetP2PAttribute(&result1, hipDevP2PAttrAccessSupported, device, peerDevice);
  hipDeviceGetP2PAttribute(&result1, hipDevP2PAttrNativeAtomicSupported, device, peerDevice);
#if (CUDA_VERSION <= 10000)
  hipDeviceGetP2PAttribute(&result1, hipDevP2PAttrHipArrayAccessSupported, device, peerDevice);
#else
  hipDeviceGetP2PAttribute(&result1, hipDevP2PAttrHipArrayAccessSupported, device, peerDevice);
#endif
#if (CUDA_VERSION != 9020)
  hipDeviceGetP2PAttribute(&result1, hipDevP2PAttrHipArrayAccessSupported, device, peerDevice);
#endif

  hipCtx_t context;
  unsigned int flags = hipDeviceMapHost;
  flags += hipDeviceScheduleBlockingSync;
  flags += hipDeviceScheduleSpin;

  if (hipCtxCreate(&context, flags, device) == hipSuccess) {
    return 0;
  }

  hipDevicePrimaryCtxSetFlags(device, flags);

  int active;
  hipDevicePrimaryCtxGetState(device, &flags, &active);

  hipCtxSetCacheConfig(hipFuncCachePreferShared);

  hipDeviceSetLimit(hipLimitPrintfFifoSize, 10);
  size_t printfsize;

  hipDeviceGetLimit(&printfsize, hipLimitPrintfFifoSize);
  return 0;
}

