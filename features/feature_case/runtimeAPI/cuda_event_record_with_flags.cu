// ====------ cuda_event_record_with_flags.cu------ *- CUDA -*-------------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===---------------------------------------------------------------------===//

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void simpleKernel() { printf("Hello simpleKernel\n"); }

int main() {
  hipEvent_t start, stop;
  hipStream_t stream;

  // Create a stream
  hipStreamCreate(&stream);

  // Create events
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Record start event with flags
  hipEventRecordWithFlags(start, stream, hipEventRecordDefault);

  // Launch a simple kernel in the stream
  simpleKernel<<<1, 1, 0, stream>>>();
  hipDeviceSynchronize();

  // Record stop event with flags
  hipEventRecordWithFlags(stop, stream, hipEventRecordDefault);

  // Wait for the event to complete
  hipEventSynchronize(stop);

  // Calculate elapsed time
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << "Elapsed time: " << milliseconds << " ms" << std::endl;

  // Clean up
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipStreamDestroy(stream);

  return 0;
}
